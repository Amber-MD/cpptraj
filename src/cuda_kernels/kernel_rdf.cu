#include "kernel_rdf.cuh"
#include "core_kernels.cuh"
#include "../CpptrajStdio.h"
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

#define BLOCKDIM 32

static inline int calc_nblocks(int ntotal, int nthreadsPerBlock)
{
  int nblocks = ntotal / nthreadsPerBlock;
  if ( (ntotal % nthreadsPerBlock) != 0 )
    nblocks++;
  return nblocks;
}

/** Calculate distances between pairs of atoms and bin them into a 1D histogram. */
int Cpptraj_GPU_RDF(unsigned long* bins, int nbins, double maximum2, double one_over_spacing,
                     const double* xyz1, int N1,
                     const double* xyz2, int N2,
                     ImageOption::Type imageType,
                     const double* box, const double* ucell, const double* recip)
{
  int* device_rdf;
  hipMalloc(((void**)(&device_rdf)), nbins * sizeof(int));
  hipMemset( &device_rdf, 0, nbins*sizeof(int) );

  double* device_xyz1;
  hipMalloc(((void**)(&device_xyz1)), N1 * 3 * sizeof(double));
  hipMemcpy(device_xyz1, xyz1, N1 * 3 * sizeof(double), hipMemcpyHostToDevice);

  double* device_xyz2;
  hipMalloc(((void**)(&device_xyz2)), N2 * 3 * sizeof(double));
  hipMemcpy(device_xyz2, xyz2, N2 * 3 * sizeof(double), hipMemcpyHostToDevice);

  double *boxDev;
  double *ucellDev, *recipDev;
  if (imageType == ImageOption::ORTHO) {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  } else if (imageType == ImageOption::NONORTHO) {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }

  // Determine number of blocks
  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  dim3 numBlocks(calc_nblocks(N1, threadsPerBlock.x), calc_nblocks(N2, threadsPerBlock.y));
  mprintf("#Atoms = %i, %i; Threads per block = %i, %i;  #Blocks = %i, %i\n",
          N1, N2, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);

  // Launch kernel
  switch (imageType) {
    case ImageOption::NONORTHO:
      kBinDistances_nonOverlap_nonOrtho<<<numBlocks, threadsPerBlock>>>(
        device_rdf, device_xyz1, N1, device_xyz2, N2, recipDev, ucellDev, maximum2, one_over_spacing);
      break;
    default:
      mprinterr("Internal Error: kernel_rdf: Unhandled image type.\n");
      return 1;
  }
  // Error check
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    mprintf("CUDA Error: %s\n", hipGetErrorString(err));
    mprinterr("CUDA Error: %s\n", hipGetErrorString(err));
    //return 1;
  }

  // Copy the result back
  int* local_bins = new int[ nbins ];
  hipMemcpy(local_bins, device_rdf, nbins*sizeof(int), hipMemcpyDeviceToHost);
  for (int ibin = 0; ibin != nbins; ibin++) {
    //mprintf("DEBUG:\tBin %i = %i (%i)\n", ibin, local_bins[ibin], device_rdf[ibin]);
    //mprintf("DEBUG:\tBin %i = %i\n", ibin, local_bins[ibin]);
    bins[ibin] += local_bins[ibin];
  }
  delete[] local_bins;
  // Free device memory
  hipFree(device_rdf);
  hipFree(device_xyz1);
  hipFree(device_xyz2);
  if (imageType == ImageOption::ORTHO)
    hipFree(boxDev);
  else if (imageType == ImageOption::NONORTHO) {
    hipFree(ucellDev);
    hipFree(recipDev);
  } 
  return 0;
}
