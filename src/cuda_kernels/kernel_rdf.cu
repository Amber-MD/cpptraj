#include "kernel_rdf.cuh"
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

#define BLOCKDIM 512

/** Calculate distances between pairs of atoms and bin them into a 1D histogram. */
void Cpptraj_GPU_RDF(int* bins,
                     const double* xyz1, int N1,
                     const double* xyz2, int N2,
                     ImageOption::Type imageType,
                     const double* box, const double* ucell, const double* recip)
{
  double* device_xyz1;
  hipMalloc(((void**)(&device_xyz1)), N1 * 3 * sizeof(double));

  double* device_xyz2;
  hipMalloc(((void**)(&device_xyz2)), N2 * 3 * sizeof(double));

  double *boxDev;
  double *ucellDev, *recipDev;
  if (imageType == ImageOption::ORTHO) {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  } else if (imageType == ImageOption::NONORTHO) {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }
}
