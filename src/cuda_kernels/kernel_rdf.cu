#include "kernel_rdf.cuh"
#include "../CpptrajStdio.h"
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

#define BLOCKDIM 32

static inline int calc_nblocks(int ntotal, int nthreadsPerBlock)
{
  int nblocks = ntotal / nthreadsPerBlock;
  if ( (ntotal % nthreadsPerBlock) != 0 )
    nblocks++;
  return nblocks;
}

/** Calculate distances between pairs of atoms and bin them into a 1D histogram. */
void Cpptraj_GPU_RDF(unsigned long* bins,
                     const double* xyz1, int N1,
                     const double* xyz2, int N2,
                     ImageOption::Type imageType,
                     const double* box, const double* ucell, const double* recip)
{
  double* device_xyz1;
  hipMalloc(((void**)(&device_xyz1)), N1 * 3 * sizeof(double));

  double* device_xyz2;
  hipMalloc(((void**)(&device_xyz2)), N2 * 3 * sizeof(double));

  double *boxDev;
  double *ucellDev, *recipDev;
  if (imageType == ImageOption::ORTHO) {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  } else if (imageType == ImageOption::NONORTHO) {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }

  // Determine number of blocks
  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  dim3 numBlocks(calc_nblocks(N1, threadsPerBlock.x), calc_nblocks(N2, threadsPerBlock.y));
  mprintf("#Atoms = %i, %i; Threads per block = %i, %i;  #Blocks = %i, %i\n",
          N1, N2, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);

}
