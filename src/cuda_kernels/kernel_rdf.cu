#include "kernel_rdf.cuh"
#include "core_kernels.cuh"
#include "../CpptrajStdio.h"
#include "../Gpu.h"
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

static inline int calc_nblocks(int ntotal, int nthreadsPerBlock)
{
  int nblocks = ntotal / nthreadsPerBlock;
  if ( (ntotal % nthreadsPerBlock) != 0 )
    nblocks++;
  return nblocks;
}

/** Report any cuda errors. */
static inline int Cuda_check(hipError_t err, const char* desc) {
  //hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    mprintf("Warning: CUDA Error %s: %s\n", desc, hipGetErrorString(err));
    mprinterr("Error: CUDA Error %s: %s\n", desc, hipGetErrorString(err));
    //return 1;
  }
  return 0;
}

/** Calculate distances between pairs of atoms and bin them into a 1D histogram. */
int Cpptraj_GPU_RDF(unsigned long* bins, int nbins, double maximum2, double one_over_spacing,
                     const double* xyz1, int N1,
                     const double* xyz2, int N2,
                     ImageOption::Type imageType,
                     const double* box, const double* ucell, const double* recip)
{
  int* device_rdf;
  Cuda_check(hipMalloc(((void**)(&device_rdf)), nbins * sizeof(int)), "Allocating rdf bins");
  Cuda_check(hipMemset( device_rdf, 0, nbins*sizeof(int) ), "Setting rdf bins to 0");

  double* device_xyz1;
  Cuda_check(hipMalloc(((void**)(&device_xyz1)), N1 * 3 * sizeof(double)), "Allocating xyz1");
  Cuda_check(hipMemcpy(device_xyz1, xyz1, N1 * 3 * sizeof(double), hipMemcpyHostToDevice), "Copying xyz1");

  double* device_xyz2;
  Cuda_check(hipMalloc(((void**)(&device_xyz2)), N2 * 3 * sizeof(double)), "Allocating xyz2");
  Cuda_check(hipMemcpy(device_xyz2, xyz2, N2 * 3 * sizeof(double), hipMemcpyHostToDevice), "Copying xyz2");

  double *boxDev;
  double *ucellDev, *recipDev;
  if (imageType == ImageOption::ORTHO) {
    Cuda_check(hipMalloc(((void**)(&boxDev)), 3 * sizeof(double)), "Allocating box");
    Cuda_check(hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice), "Copying box");
  } else if (imageType == ImageOption::NONORTHO) {
    Cuda_check(hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double)), "Allocating ucell");
    Cuda_check(hipMalloc(((void**)(&recipDev)), 9 * sizeof(double)), "Allocating frac");
    Cuda_check(hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice), "Copying ucell");
    Cuda_check(hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice), "Copying frac");
  }

  // Determine number of blocks
  unsigned int BLOCKDIM = CpptrajGpu::MaxBlockDim_2D();

  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  dim3 numBlocks(calc_nblocks(N1, threadsPerBlock.x), calc_nblocks(N2, threadsPerBlock.y));
  mprintf("#Atoms = %i, %i; Threads per block = %i, %i;  #Blocks = %i, %i\n",
          N1, N2, threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);

  // Launch kernel
  switch (imageType) {
    case ImageOption::NONORTHO:
      kBinDistances_nonOverlap_nonOrtho<<<numBlocks, threadsPerBlock>>>(
        device_rdf, device_xyz1, N1, device_xyz2, N2, recipDev, ucellDev, maximum2, one_over_spacing);
      break;
    default:
      mprinterr("Internal Error: kernel_rdf: Unhandled image type.\n");
      return 1;
  }
  // Error check
  Cuda_check(hipGetLastError(), "kernel launch");

  // Copy the result back
  int* local_bins = new int[ nbins ];
  hipMemcpy(local_bins, device_rdf, nbins*sizeof(int), hipMemcpyDeviceToHost);
  for (int ibin = 0; ibin != nbins; ibin++) {
    //mprintf("DEBUG:\tBin %i = %i (%i)\n", ibin, local_bins[ibin], device_rdf[ibin]);
    //mprintf("DEBUG:\tBin %i = %i\n", ibin, local_bins[ibin]);
    bins[ibin] += local_bins[ibin];
  }
  delete[] local_bins;
  // Free device memory
  hipFree(device_rdf);
  hipFree(device_xyz1);
  hipFree(device_xyz2);
  if (imageType == ImageOption::ORTHO)
    hipFree(boxDev);
  else if (imageType == ImageOption::NONORTHO) {
    hipFree(ucellDev);
    hipFree(recipDev);
  } 
  return 0;
}
