#include "GistCudaSetup.cuh"
#include "GistCudaCalc.cuh"

#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif

/**
 * Allocate memory on the GPU.
 * @param array: The pointer to the array, which will be allocated on the GPU.
 * @param size: An integer giving the size of the array, which will be allocated.
 * @throws: CudaException if a problem occurs.
 */
__host__
void allocateCuda(void **array, int size) {
  // Check if the array is actually free, if not, it will be freed 
  // (fun fact: checking is not necessary, one could also simply free the memory).
  if ((*array) != NULL) {
    hipFree(*array);
  }
  // If something goes wrong, throw exception
  if (hipMalloc(array, size) != hipSuccess) {
    throw CudaException();
  }
}

/**
 * Copy memory from the CPU to the GPU.
 * @param array: The array from which the values shall be copied.
 * @param array_c: The array on the device, to which the values shall be copied.
 * @param size: The size of the stuff which will be copied.
 * @throws: CudaException if something goes wrong.
 */
__host__
void copyMemoryToDevice(void *array, void *array_c, int size) {
  // If something goes wrong, throw exception
  // In this case only copying can go wrong.
  if (hipMemcpy(array_c, array, size, hipMemcpyHostToDevice) != hipSuccess) {
    throw CudaException();
  }
}

/**
 * A simple helper function that copies a lot of stuff to the GPU (as structs).
 * @param charge: An array holding the charges for the different atoms.
 * @param atomtype: An array holding the integers for the atom types of the different atoms.
 * @param solvent: An array of boolean values, holding the information whether a certain atom is solvent or solute.
 * @param atomNumber: The total number of atoms.
 * @param atomProps_c: A pointer to an array on the GPU, which will hold the atom properties.
 * @param ljA: An array holding the lennard-jones parameter A for each atom type pair.
 * @param ljB: An array holding the lennard-jones parameter B for each atom type pair.
 * @param length: The length of the two aforementioned arrays (ljA & ljB).
 * @param lJparams_c: A pointer to an array on the GPU, which will hold the lj parameters.
 * @throws: CudaException if something bad happens.
 */
__host__
void copyMemoryToDeviceStruct(float *charge, int *atomtype, bool *solvent, int *molecule, int atomNumber, void **atomProps_c,
                              float *ljA, float *ljB, int length, void **lJparams_c) {
  // Check if the two arrays are free. Again, this could be removed (but will stay!)
  if ((*atomProps_c) != NULL) {
    hipFree(*atomProps_c);
  }
  if ((*lJparams_c) != NULL) {
    hipFree(*lJparams_c);
  }
  // Allocate the necessary memory on the GPU.
  if (hipMalloc(atomProps_c, atomNumber * sizeof(AtomProperties)) != hipSuccess) {
    throw CudaException();
  }
  if (hipMalloc(lJparams_c, length * sizeof(ParamsLJ)) != hipSuccess) {
    throw CudaException();
  }

  // Create an array for the lennard-jones parameters.
  ParamsLJ *ljp = (ParamsLJ *) malloc (length * sizeof(ParamsLJ));
  // Add the lennard-jones parameters to the array.
  for (int i = 0; i < length; ++i) {
    ljp[i] = ParamsLJ(ljA[i], ljB[i]);
  }

  // Create an array for the atom properties.
  AtomProperties *array = (AtomProperties *)malloc(atomNumber * sizeof(AtomProperties));
  // Add the properties into the array.
  for (int i = 0; i < atomNumber; ++i) {
    array[i] = AtomProperties(charge[i], atomtype[i], solvent[i], molecule[i]);
  }
  // Copy the memory from the host to the device.
  if (hipMemcpy((*atomProps_c), array, atomNumber * sizeof(AtomProperties), hipMemcpyHostToDevice) != hipSuccess) {
    throw CudaException();
  }
  if (hipMemcpy((*lJparams_c), ljp, length * sizeof(ParamsLJ), hipMemcpyHostToDevice) != hipSuccess) {
    throw CudaException();
  }

  // Free the two arrays (so that no memory leak occurs).
  free(ljp);
  free(array);
}

/**
 * Free an array on the CUDA capable device.
 * @param array: The array you want to free.
 */
__host__
void freeCuda(void *array) {
  hipFree(array);
}


// This is coded C-like, but uses exceptions.
/**
 * This starts the cuda kernel, thus it is actually a quite long function.
 */
__host__
std::vector<std::vector<float> > doActionCudaEnergy(const double *coords, int *NBindex_c, int ntypes, void *parameter, void *molecule_c,
                            int boxinfo, float *recip_o_box, float *ucell, int maxAtoms, float *min_c, float *max_c, int headAtomType, 
                            float neighbourCut2, int *result_o, int *result_n, float *result_w_c, float *result_s_c,
                            int *result_O_c, int *result_N_c, bool doorder) {
  Coordinates *coords_c   = NULL;
  float *recip_b_c  = NULL;
  float *ucell_c    = NULL;
  
  

  float *result_A = (float *) calloc(maxAtoms, sizeof(float));
  float *result_s = (float *) calloc(maxAtoms, sizeof(float));
  // TODO: Fix this, test is actually a quite bad name here!
  Coordinates *coord_array = (Coordinates *) calloc(maxAtoms, sizeof(Coordinates));
  
  // Casting
  AtomProperties *sender = (AtomProperties *) molecule_c;
  ParamsLJ *lennardJonesParams = (ParamsLJ *) parameter;
  
  // Create Boxinfo and Unit cell. This is actually very important for the speed (otherwise
  // there would be LOTS of access to non-local variables).
  BoxInfo boxinf;
  if (boxinfo != 0) {
    boxinf = BoxInfo(recip_o_box, boxinfo);
  }
  UnitCell ucellN;
  if (boxinfo == 2) {
    ucellN = UnitCell(ucell);
  }
  
  // Add the coordinates to the array.
  // TODO: Fix Test here also!
  for (int i = 0; i < maxAtoms; ++i) {
    coord_array[i] = Coordinates(&coords[i * 3]);
  }

  // vectors that will return the necessary information.
  std::vector<std::vector<float> > result;
  std::vector<float> result_esw;
  std::vector<float> result_eww;

  // Allocate space on the GPU
  if (hipMalloc(&coords_c, maxAtoms * sizeof(Coordinates)) != hipSuccess) {
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }


  // Copy the data to the GPU
  if (hipMemcpy(coords_c, coord_array, maxAtoms * sizeof(Coordinates), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }
  if (hipMemcpy(result_w_c, result_A, maxAtoms * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }
  if (hipMemcpy(result_s_c, result_s, maxAtoms * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }

  // If the doorder calculation is used, it needs to calculate everything differently, so the slow version is used
  // (this is about 10% slower).
  if (doorder) {
    cudaCalcEnergySlow<<< (maxAtoms + SLOW_BLOCKSIZE) / SLOW_BLOCKSIZE, SLOW_BLOCKSIZE >>> (coords_c, NBindex_c, ntypes, lennardJonesParams, sender,
                                                                                            boxinf, ucellN, maxAtoms, result_w_c, result_s_c, min_c, max_c,
                                                                                            headAtomType, neighbourCut2, result_O_c, result_N_c);
  } else {
    // Uses a 2D array, which is nice for memory access.
    dim3 threadsPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 numBlocks((maxAtoms + threadsPerBlock.x) / threadsPerBlock.x, (maxAtoms + threadsPerBlock.y) / threadsPerBlock.y);
    // The actual call of the device function
    cudaCalcEnergy<<<numBlocks, threadsPerBlock>>> (coords_c, NBindex_c, ntypes, lennardJonesParams, sender,
                                                                      boxinf, ucellN, maxAtoms, result_w_c, result_s_c, min_c, max_c,
                                                                      headAtomType, neighbourCut2, result_O_c, result_N_c);
    // Check if there was an error.
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
      printf("returned %s\n", hipGetErrorString(hipError_t));
    }
  }
  // Return the results of the calculation to the main memory
  if (hipMemcpy(result_A, result_w_c, maxAtoms * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }  
  

  if (hipMemcpy(result_s, result_s_c, maxAtoms * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }


  
  if (hipMemcpy(result_o, result_O_c, maxAtoms * 4 * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }
  
  if (hipMemcpy(result_n, result_N_c, maxAtoms * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
    hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
    free(result_A); free(result_s); free(coord_array);
    throw CudaException();
  }

  for (int i = 0; i < maxAtoms; ++i) {
    result_eww.push_back(result_A[i]);
    result_esw.push_back(result_s[i]);
  }

  result.push_back(result_eww);
  result.push_back(result_esw);

  // Free everything used in here.
  hipFree(coords_c); hipFree(recip_b_c); hipFree(ucell_c);
  free(result_A); free(result_s); free(coord_array);
  
  return result;
}
