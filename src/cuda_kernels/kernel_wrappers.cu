#include "hip/hip_runtime.h"
#include "kernel_wrappers.cuh"
#ifdef DEBUG_CUDA
#include <cstdio>
#endif
#if defined(__HIP_PLATFORM_HCC__)
#include <hip/hip_runtime.h>
#include "../HipDefinitions.h"
#endif
#include "core_kernels.cuh"

#define BLOCKDIM 512

/** Calculate the closest distances between atoms in solvent molecules and
  * the given point.
  * \param SolventMols_ Coordinates for each atom of each solvent molecule.
  * \param D_ Output distances for each molecule.
  * \param maskCenter Point to calculate distances to.
  * \param maxD Maximum possible distance.
  * \param NMols Number of solvent molecules.
  * \param NAtoms Number of atoms in each solvent molecule.
  * \param type Imaging type (none, ortho, non-ortho).
  * \param box Box coordinates.
  * \param ucell Unit cell matrix.
  * \param recip Fractional cell matrix.
  */
void Action_Closest_Center(const double *SolventMols_, double *D_, const double* maskCenter,
                           double maxD, int NMols, int NAtoms, ImageOption::Type type,
                           const double* box, const double* ucell, const double* recip)
{
  #ifdef DEBUG_CUDA
  hipEvent_t start_event, stop_event;
  float time_gpu;
  #endif
  double *devI2Ptr;
  double *devI1Ptr;
  double *devO1Ptr;
  double *boxDev;
  double *ucellDev, *recipDev;

  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));

  hipMalloc(((void **)(&devI1Ptr)),3 * sizeof(double ));
  hipMemcpy(devI1Ptr,maskCenter,3 * sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);

  if (type == ImageOption::ORTHO)
  {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  }
  if (type == ImageOption::NONORTHO)
  {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }

  int active_size  =  BLOCKDIM/NAtoms * NAtoms;
  int NBlocks = ceil(float(NMols)/ (BLOCKDIM));

  dim3 dimGrid0 = dim3(NBlocks,1);
  dim3 dimBlock0 = dim3(BLOCKDIM,1);

  #ifdef DEBUG_CUDA
  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms);
  printf("active_size =  %d\n", active_size);
  printf("NBlocks =  %d\n", NBlocks);
  printf("sizeof(double) = %d\n", sizeof(double));
  printf("About to launch kernel.\n");

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  #endif

  switch (type) {
    case ImageOption::NO_IMAGE:
      kClosestDistsToPt_NoImage<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD, NMols, NAtoms,active_size);
      break;
    case ImageOption::ORTHO:
      kClosestDistsToPt_Ortho<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD,boxDev, NMols, NAtoms,active_size);
      break;
    case ImageOption::NONORTHO:
      kClosestDistsToPt_Nonortho<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD,ucellDev, recipDev, NMols, NAtoms,active_size);
  }

  //hipDeviceSynchronize();

  #ifdef DEBUG_CUDA
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&time_gpu,start_event, stop_event );

  printf("Done with kernel CUDA Kernel Time: %.2f\n", time_gpu);
  #endif

  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
  if (type == ImageOption::ORTHO)
    hipFree(boxDev);
  if (type == ImageOption::NONORTHO)
  {
    hipFree(ucellDev);
    hipFree(recipDev);
  }
}

/** Calculate the closest distances between atoms in solvent molecules and
  * each solute atom.
  * \param SolventMols_ Coordinates for each atom of each solvent molecule.
  * \param D_ Output distances for each molecule.
  * \param Solute_atoms Coordinates for each solute atom.
  * \param maxD Maximum possible distance.
  * \param NMols Number of solvent molecules.
  * \param NAtoms Number of atoms in each solvent molecule.
  * \param NSAtoms Number of solute atoms.
  * \param type Imaging type (none, ortho, non-ortho).
  * \param box Box coordinates.
  * \param ucell Unit cell matrix.
  * \param recip Fractional cell matrix.
  */
void Action_Closest_NoCenter(const double *SolventMols_, double *D_, const double *Solute_atoms,
                             double maxD, int NMols, int NAtoms, int NSAtoms, ImageOption::Type type,
                             const double* box, const double* ucell, const double* recip)
{
  #ifdef DEBUG_CUDA
  hipEvent_t start_event, stop_event;
  float time_gpu;
  #endif
  double *devI3Ptr;
  double *devI2Ptr;
  double *devO1Ptr;
  double *boxDev;
  double *ucellDev, *recipDev;

  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));

  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&devI3Ptr)), NSAtoms * 3 * sizeof(double ));
  hipMemcpy(devI3Ptr,Solute_atoms,NSAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);

  if (type == ImageOption::ORTHO)
  {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  }
  if (type == ImageOption::NONORTHO)
  {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }

  int active_size  =  BLOCKDIM/NAtoms * NAtoms;
  int NBlocks =  ceil(NMols * NAtoms / float(active_size));

  dim3 dimGrid0 = dim3(NBlocks,1);
  dim3 dimBlock0 = dim3(BLOCKDIM,1);

  #ifdef DEBUG_CUDA
  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms);
  printf("active_size =  %d\n", active_size);
  printf("NBlocks =  %d\n", NBlocks);
  printf("sizeof(double) = %d\n", sizeof(double));
  printf("About to launch kernel.\n");

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  #endif

  switch (type) {
    case ImageOption::NO_IMAGE:
      kClosestDistsToAtoms_NoImage<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, NMols, NAtoms,NSAtoms,active_size);
      break;
    case ImageOption::ORTHO:
      kClosestDistsToAtoms_Ortho<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, boxDev,  NMols, NAtoms,NSAtoms,active_size);
      break;
    case ImageOption::NONORTHO:
      kClosestDistsToAtoms_Nonortho<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, ucellDev, recipDev,  NMols, NAtoms,NSAtoms,active_size);
    break;
  }
  //hipDeviceSynchronize();

  #ifdef DEBUG_CUDA
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&time_gpu,start_event, stop_event );

  printf("Done with kernel CUDA Kernel Time: %.2f\n", time_gpu);
  #endif

  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI2Ptr);
  hipFree(devI3Ptr);
  if (type == ImageOption::ORTHO)
    hipFree(boxDev);
  if (type == ImageOption::NONORTHO)
  {
    hipFree(ucellDev);
    hipFree(recipDev);
  }
}
