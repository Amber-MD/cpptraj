#include "hip/hip_runtime.h"
#include "GistCudaCalc.cuh"

#include <cstdio>

#define ELECTOAMBER_2 332.05221729


 /**
  * Calculate the squared distance in an orthorhombic box. See cpptraj implementation.
  * @param vec1: The first point of the distance calculation.
  * @param vec2: The seconf point of the distance calculation
  * @param box: The boxinfo of the object.
  * @return: The minimal distance in an orthorhombic box.
  */
__device__ 
float dist2_imageOrtho(float *vec1, float *vec2, BoxInfo box) {
  if (box[0] == 0 || box[1] == 0 || box[2] == 0) {
    return -1;
  }
  float x = abs(vec1[0] - vec2[0]);
  float y = abs(vec1[1] - vec2[1]);
  float z = abs(vec1[2] - vec2[2]);

  while (x > box[0]) {
    x -= box[0];
  }
  while (y > box[1]) {
    y -= box[1];
  }
  while (z > box[2]) {
    z -= box[2];
  }

  x = min(x, box[0] - x);
  y = min(y, box[1] - y);
  z = min(z, box[2] - z);

  return x * x + y * y + z * z;
}

/**
 * Calculate M * v.
 * @param vec: The vector v.
 * @param mat3x3: The matrix M.
 * @param ret: The values to be returned. If null, returns into vec.
 */
__device__
void scalarProd(float* vec, BoxInfo mat3x3, float *ret) {
  if (ret != NULL) {
    ret[0] = vec[0] * mat3x3[0] + vec[1] * mat3x3[1] + vec[2] * mat3x3[2];
    ret[1] = vec[0] * mat3x3[3] + vec[1] * mat3x3[4] + vec[2] * mat3x3[5];
    ret[2] = vec[0] * mat3x3[6] + vec[1] * mat3x3[7] + vec[2] * mat3x3[8];
  } else {
    float x = vec[0];
    float y = vec[1];
    float z = vec[2];
    vec[0] = x * mat3x3[0] + y * mat3x3[1] + z * mat3x3[2];
    vec[1] = x * mat3x3[3] + y * mat3x3[4] + z * mat3x3[5];
    vec[2] = x * mat3x3[6] + y * mat3x3[7] + z * mat3x3[8];
  }
}

/**
 * Find the result_O squared distance in a non-orthogonal box.
 * @param vec1: Position vector of point 1.
 * @param vec2: Position vector of point 2.
 * @param recip: The inverse of the ucell.
 * @param ucell: The box matrix.
 * @return: The minimal squared distance between two atoms, also considering the images.
 */
__device__
float dist2_imageNonOrtho(float *vec1, float *vec2, BoxInfo recip, UnitCell ucell) {
  float vecRecip1[3];
  float vecRecip2[3];
  scalarProd(vec1, recip, vecRecip1);
  scalarProd(vec2, recip, vecRecip2);
  float r_2 = dist2_imageNonOrthoRecip(vecRecip1, vecRecip2, ucell);
  
  return r_2;
}

/**
 * Calculate if the distance to an image is smaller than a given distance.
 * @param f: The first vector in the reciprocal space.
 * @param vec2Cartesian: The second vector in cartesian coordinates.
 * @param nx: Which neighbour in x direction?
 * @param ny: Which neighbour in y direction?
 * @param nz: Which neighbour in z direction?
 * @param ucell: The box matrix.
 * @param finalMin: The already calculated minimum.
 * @return: The new minimum, if it is smaller than finalMin, finalMin otherwise.
 */
__device__
float calcIfDistIsSmaller(float *f, float *vec2Cartesian, int nx, int ny, int nz, UnitCell ucell, float finalMin) {
  float fx = f[0] + nx;
  float fy = f[1] + ny;
  float fz = f[2] + nz;
  // Bring f back in Cartesian coordinates
  float x = fx * ucell[0] + fy * ucell[3] + fz * ucell[6];
  float y = fx * ucell[1] + fy * ucell[4] + fz * ucell[7];
  float z = fx * ucell[2] + fy * ucell[5] + fz * ucell[8];
  x -= vec2Cartesian[0];
  y -= vec2Cartesian[1];
  z -= vec2Cartesian[2];
  float min = x * x + y * y + z * z;
  if ( min < finalMin || finalMin < 0) {
    return min;
  }
  return finalMin;
}

/**
 * Calculate the distance in a non-orthorhombic box, if the vectors are already
 * multiplied by the inverse box matrix.
 * @param vec1: The first position vector.
 * @param vec2: The second position vector.
 * @param ucell: The box cell.
 * @return: The minimal distance between images.
 */
__device__
float dist2_imageNonOrthoRecip(float * vec1, float * vec2, UnitCell ucell) {
    
  // Bring the points back into the main unit cell
  float fx = vec1[0] - floor(vec1[0]);
  float fy = vec1[1] - floor(vec1[1]);
  float fz = vec1[2] - floor(vec1[2]);
  float f2x = vec2[0] - floor(vec2[0]);
  float f2y = vec2[1] - floor(vec2[1]);
  float f2z = vec2[2] - floor(vec2[2]);

  float vec[3] = {fx, fy, fz};

  // Bring f2 back in cartesian coordinates
  float xFactor = f2x * ucell[0] + f2y * ucell[3] + f2z * ucell[6];
  float yFactor = f2x * ucell[1] + f2y * ucell[4] + f2z * ucell[7];
  float zFactor = f2x * ucell[2] + f2y * ucell[5] + f2z * ucell[8];
  float vec2Real[3] = {xFactor, yFactor, zFactor};

  // Now the different cases, and always store the minimum
  // Define the finalMinimum as a negative value, since it can never
  // actually be negative this is fine.
  float finalMinimum = -1.0;

  // Run through all cells
  for (int ix = -1; ix <= 1; ++ix) {
    for (int iy = -1; iy <= 1; ++iy) {
      for (int iz = -1; iz <= 1; ++iz) {
        finalMinimum = calcIfDistIsSmaller(vec, vec2Real, ix, iy, iz, ucell, finalMinimum);
      }
    }
  }

  return finalMinimum;
}

/**
 * Calculates the distance between two vectors, without imaging.
 * @param vec1: The position vector of the first atom.
 * @param vec2: The position vector of the second atom.
 * @return: The squared distance between the two positions.
 */
__device__
float dist2_noImage(float *vec1, float *vec2) {
  float x = vec1[0] - vec2[0];
  float y = vec1[1] - vec2[1];
  float z = vec1[2] - vec2[2];

  return x*x + y*y + z*z;
}

/**
 * Caclulate the total energy between two atoms.
 * @param vec1: The position vector of atom 1.
 * @param vec2: The position vector of atom 2.
 * @param q1: The charge of atom 1.
 * @param q2: The charge of atom 2.
 * @param LJA: The lennard jones A parameter.
 * @param LJB: The lennard jones B parameter.
 * @param boxinfo: Which kind of box, 0 not periodic, 1 orthorhombic, 2 otherwise.
 * @param recip_o_box: Holds either the inverse of the cell matrix, if boxinfo is 2,
 *                        or the box dimensions, if boxinfo is 1, or is NULL, if boxinfo is 0.
 * @param ucell: Holds the cell matrix, if boxinfo is 2, NULL otherwise.
 * @return: The total interaction energy between the two atoms.
 */
__device__
float calcTotalEnergy(float q1, float q2, 
                            float LJA, float LJB, float r_2) {
#ifdef DEBUG_GIST_CUDA
  if (r_2 <= 0.000001 && r_2 >= -0.000001) {
    printf("(%8.3f, %8.3f, %8.3f) (%8.3f, %8.3f, %8.3f) %d\n", vec1[0], vec1[1], vec1[2], vec2[0], vec2[1], vec2[2], boxinfo);
  }
#endif
  return calcVdWEnergy(r_2, LJA, LJB) + calcElectrostaticEnergy(r_2, q1, q2);
}

/**
 * Calculate the distance between two different points.
 * @param vec1: The first vector to calculate the distance.
 * @param vec2: The second vector to calculate the distance.
 * @param recip_o_box: The boxinfo, either the box or the reciprocal.
 * @param ucell: The unitcell of a box.
 * @return: The squared distance between two points.
 */
__device__
float calcDist(float *vec1, float *vec2, BoxInfo recip_o_box,
                    UnitCell ucell) {
  float r_2 = 0;
  switch(recip_o_box.boxinfo) {
    case 0:
      r_2 = dist2_noImage(vec1, vec2);
      break;
    case 1:
      // Uses recip for box info as well;
      r_2 = dist2_imageOrtho(vec1, vec2, recip_o_box);
      break;
    case 2:
      r_2 = dist2_imageNonOrtho(vec1, vec2, recip_o_box, ucell);
      break;
    default:
      r_2 = 0;
  }
  return r_2;
}

/**
 * Calculate the Van der Waals energy between two atoms.
 * @param r_2: The squared distance between the two atoms.
 * @param LJA: The A part of the lennard jones potential.
 * @param LJB: The B part if the Lennard Jones potential.
 * @return: The Van der Waals energy.
 */
__device__
float calcVdWEnergy(float r_2, float LJA, float LJB) {
  float r_6 = r_2 * r_2 * r_2;
  float r_12 = r_6 * r_6;
  float LJ =  LJA / r_12 - LJB / r_6;
  return LJ;
}

/**
 * Calculate the electrostatic energy between two different atoms.
 * @param r_2: The square distance between the two atoms.
 * @param q1: The charge of atom 1.
 * @param q2: The charge of atom 2.
 * @return: The electrostatic energy between the two atoms.
 */
__device__
float calcElectrostaticEnergy(float r_2, float q1, float q2) {
  double charge = q1 * q2 * ELECTOAMBER_2;
  double r = sqrt(r_2);
  float ELE = charge / r;
  return ELE;
}

/**
 * Get the index into the lennard jones parameter array.
 * @param a1: The atom type index of atom 1.
 * @param a2: The atom type index of atom 2.
 * @param NBindex: The arrays holding the indices into the LJ array.
 * @param ntypes: The number of atom types.
 * @return: The index into the parameter arrays.
 */
__device__
int getLJIndex(int a1, int a2, int *NBindex, int ntypes) {
  return NBindex[a1 * ntypes + a2];
}

/**
 * Get the LJ parameters from a parameter array.
 * @param a1: The atom type index of atom 1.
 * @param a2: The atom type index of atom 2.
 * @param NBindex: The indices into the parameter array.
 * @param ntypes: The number of atom types.
 * @param paramsLJ: The parameter array.
 * @return: The LJ parameter belonging to the atom type pair a1, a2.
 */
__device__
ParamsLJ getLJParam(int a1, int a2, int *NBindex, int ntypes, ParamsLJ *paramsLJ) {
  int idx = getLJIndex(a1, a2, NBindex, ntypes);
  if (idx < 0) {
    return ParamsLJ();
  }
  return paramsLJ[idx];
}

/**
 * Checks if atom is on grid.
 * @param vec: Position vector to the atom.
 * @param min: The grid starting position.
 * @param max: The end of the grid.
 * @return: True if vector is on grid.
 */
__device__
bool isOnGrid(float *vec, float *min, float *max) {
  return ( ( (vec[0] >= min[0]) && (vec[0] <= max[0]) ) &&
           ( (vec[1] >= min[1]) && (vec[1] <= max[1]) ) &&
           ( (vec[2] >= min[2]) && (vec[2] <= max[2]) ) );
}

/**
 * Calculate the energy on the GPU.
 * @param coords: An array holding all the coordinates of all atoms.
 * @param NBindex: An array holding indices into the LJ parameter arrays.
 * @param ntypes: The number of atom types.
 * @param paramsLJA: The A LJ parameters.
 * @param paramsLJB: The B LJ parameters.
 * @param charges: The charges of the atoms.
 * @param boxinfo: Which kind of box, 0 not periodic, 1 orthorhombic, 2 otherwise.
 * @param recip_o_box: Holds either the inverse of the cell matrix, if boxinfo is 2,
 *                        or the box dimensions, if boxinfo is 1, or is NULL, if boxinfo is 0.
 * @param ucell: Holds the cell matrix, if boxinfo is 2, NULL otherwise.
 * @param maxAtoms: The number of atoms in the system.
 * @param a_types: The different atom types of the atoms.
 * @param solvent: True if atom is a solvent atom, false otherwise.
 * @param molecule: The number of the molecule this atom belong to.
 * @param result_ww: The result of the water - water interactions.
 * @param result_sw: The result of the solute - water interactions.
 * @param min: The minimum values of the grid.
 * @param max: The maximum values of the grid.
 */
__global__
void cudaCalcEnergy(Coordinates *coords, int *NBindex, int ntypes, ParamsLJ *parameterLJ, AtomProperties *atomProps, 
                          BoxInfo recip_o_box, UnitCell ucell, int maxAtoms, float *result_ww, float *result_sw, 
                          float *min, float *max, int headAtomType, float neighbourCut2, int *result_O, int *result_N) {
  

  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;
  if ( (a1 >= maxAtoms) || (a2 >= maxAtoms) || (a1 == a2)) {
    return;
  }

  AtomProperties atom1 = atomProps[a1];
  AtomProperties atom2 = atomProps[a2];

  // Do not calculate if the two values are the same or they belong to the same molecule.
  if ( (atom1.molecule != atom2.molecule)) {

    Coordinates t1 = coords[a1];
    Coordinates t2 = coords[a2];
    ParamsLJ lj = getLJParam(atom1.atomType, atom2.atomType, NBindex, ntypes, parameterLJ);

    float vec1[3] = {t1.x, t1.y, t1.z};
    float vec2[3] = {t2.x, t2.y, t2.z};
    float r_2 = calcDist(vec1, vec2, recip_o_box, ucell);
    float energy = calcTotalEnergy(atom1.charge, atom2.charge, lj.A, lj.B, r_2);
    
    if (atom2.solvent) {
      atomicAdd(&(result_ww[a1]), energy * 0.5f);
    } else {
      atomicAdd(&(result_sw[a1]), energy);
    }
  
  }
}

/**
 * Calculate the energy on the GPU. This implementation is somewhat slower,
 * but is able to calculate the order parameters as well as the rest.
 * @param coords: An array holding all the coordinates of all atoms.
 * @param NBindex: An array holding indices into the LJ parameter arrays.
 * @param ntypes: The number of atom types.
 * @param paramsLJA: The A LJ parameters.
 * @param paramsLJB: The B LJ parameters.
 * @param charges: The charges of the atoms.
 * @param boxinfo: Which kind of box, 0 not periodic, 1 orthorhombic, 2 otherwise.
 * @param recip_o_box: Holds either the inverse of the cell matrix, if boxinfo is 2,
 *                        or the box dimensions, if boxinfo is 1, or is NULL, if boxinfo is 0.
 * @param ucell: Holds the cell matrix, if boxinfo is 2, NULL otherwise.
 * @param maxAtoms: The number of atoms in the system.
 * @param a_types: The different atom types of the atoms.
 * @param solvent: True if atom is a solvent atom, false otherwise.
 * @param molecule: The number of the molecule this atom belong to.
 * @param result_ww: The result of the water - water interactions.
 * @param result_sw: The result of the solute - water interactions.
 * @param min: The minimum values of the grid.
 * @param max: The maximum values of the grid.
 */
__global__
void cudaCalcEnergySlow(Coordinates *coords, int *NBindex, int ntypes, ParamsLJ *parameterLJ, AtomProperties *atomProps, 
  BoxInfo recip_o_box, UnitCell ucell, int maxAtoms, float *result_ww, float *result_sw,
  float *min, float *max, int headAtomType, float neighbourCut2, int *result_O, int *result_N) {
  
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (a1 >= maxAtoms) {
    return;
  }
  
  AtomProperties atom1 = atomProps[a1];
  float distances[4] = {HUGE_C, HUGE_C, HUGE_C, HUGE_C};
  result_N[a1] = 0;
  result_O[4 * a1 + 3] = 0;
  result_O[4 * a1 + 2] = 0;
  result_O[4 * a1 + 1] = 0;
  result_O[4 * a1    ] = 0;
  float energy_ww = 0.0f;
  float energy_sw = 0.0f;
  for (int a2 = 0; a2 < maxAtoms; ++a2) {
    AtomProperties atom2 = atomProps[a2];
    // Do not calculate if the two values are the same or they belong to the same molecule.
    if ((a1 != a2) && (atom1.molecule != atom2.molecule)) {
      Coordinates t1 = coords[a1];
      Coordinates t2 = coords[a2];
      ParamsLJ lj = getLJParam(atom1.atomType, atom2.atomType, NBindex, ntypes, parameterLJ);
      float vec1[3] = {t1.x, t1.y, t1.z};
      float vec2[3] = {t2.x, t2.y, t2.z};
      float r_2 = calcDist(vec1, vec2, recip_o_box, ucell);
      float energy = calcTotalEnergy(atom1.charge, atom2.charge, lj.A, lj.B, r_2);
      if ((atom2.atomType == headAtomType) && atom2.solvent && atom1.solvent) {
        if (r_2 < distances[0]) {
          distances[3] = distances[2];
          distances[2] = distances[1];
          distances[1] = distances[0];
          distances[0] = r_2;
          result_O[4 * a1 + 3] = result_O[4 * a1 + 2];
          result_O[4 * a1 + 2] = result_O[4 * a1 + 1];
          result_O[4 * a1 + 1] = result_O[4 * a1    ];
          result_O[4 * a1    ] = a2;
        } else if (r_2 < distances[1]) {
          distances[3] = distances[2];
          distances[2] = distances[1];
          distances[1] = r_2;
          result_O[4 * a1 + 3] = result_O[4 * a1 + 2];
          result_O[4 * a1 + 2] = result_O[4 * a1 + 1];
          result_O[4 * a1 + 1] = a2;
        } else if (r_2 < distances[2]) {
          distances[3] = distances[2];
          distances[2] = r_2;
          result_O[4 * a1 + 3] = result_O[4 * a1 + 2];
          result_O[4 * a1 + 2] = a2;
        } else if (r_2 < distances[3]) {
          distances[3] = r_2;
          result_O[4 * a1 + 3] = a2;
        }
        if (r_2 < neighbourCut2) {
          result_N[a1] += 1;
        }
      }
      if (atom2.solvent) {
        energy_ww += energy * 0.5f;
      } else {
        energy_sw += energy;
      }
    }
  }
  result_ww[a1] = energy_ww;
  result_sw[a1] = energy_sw;
}
