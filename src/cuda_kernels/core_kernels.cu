#include "hip/hip_runtime.h"
#include "core_kernels.cuh"
#define BLOCKDIM 512
#define RSIZE 512

// -----------------------------------------------------------------------------
/** \return Shortest imaged distance between given coordinates in fractional space.
  * NOTE: This function is complicated hence we will put into a __device__ only function.
  */
__device__ double NonOrtho_dist(double a0, double a1, double a2,
                                double b0, double b1, double b2,
                                const double *ucell)
{
  //int ixyz[3];
  double minIn  = -1.0;

   //double closest2
  // The floor() calls serve to bring each point back in the main unit cell.
  double fx = a0 - floor(a0);
  double fy = a1 - floor(a1);
  double fz = a2 - floor(a2); 
  double f2x = b0 - floor(b0);
  double f2y = b1 - floor(b1);
  double f2z = b2 - floor(b2);
  // f2 back in Cartesian space
  double X_factor = (f2x*ucell[0] + f2y*ucell[3] + f2z*ucell[6]);
  double Y_factor = (f2x*ucell[1] + f2y*ucell[4] + f2z*ucell[7]);
  double Z_factor = (f2x*ucell[2] + f2y*ucell[5] + f2z*ucell[8]);
  // Precompute some factors
  double fxm1 = fx - 1.0;
  double fxp1 = fx + 1.0;
  double fym1 = fy - 1.0;
  double fyp1 = fy + 1.0;
  double fzm1 = fz - 1.0;
  double fzp1 = fz + 1.0;

  double fxm1u0 = fxm1 * ucell[0];
  double fxu0   = fx   * ucell[0];
  double fxp1u0 = fxp1 * ucell[0];
  double fxm1u1 = fxm1 * ucell[1];
  double fxu1   = fx   * ucell[1];
  double fxp1u1 = fxp1 * ucell[1];
  double fxm1u2 = fxm1 * ucell[2];
  double fxu2   = fx   * ucell[2];
  double fxp1u2 = fxp1 * ucell[2];

  double fym1u3 = fym1 * ucell[3];
  double fyu3   = fy   * ucell[3];
  double fyp1u3 = fyp1 * ucell[3];
  double fym1u4 = fym1 * ucell[4];
  double fyu4   = fy   * ucell[4];
  double fyp1u4 = fyp1 * ucell[4];
  double fym1u5 = fym1 * ucell[5];
  double fyu5   = fy   * ucell[5];
  double fyp1u5 = fyp1 * ucell[5];

  double fzm1u6 = fzm1 * ucell[6];
  double fzu6   = fz   * ucell[6];
  double fzp1u6 = fzp1 * ucell[6];
  double fzm1u7 = fzm1 * ucell[7];
  double fzu7   = fz   * ucell[7];
  double fzp1u7 = fzp1 * ucell[7];
  double fzm1u8 = fzm1 * ucell[8];
  double fzu8   = fz   * ucell[8];
  double fzp1u8 = fzp1 * ucell[8];

  // Calc ix iy iz = 0 case
  double x = (fxu0 + fyu3 + fzu6) - X_factor;
  double y = (fxu1 + fyu4 + fzu7) - Y_factor;
  double z = (fxu2 + fyu5 + fzu8) - Z_factor;
  // DEBUG
  //mprintf("DEBUG: a2: %g %g %g\n",(fxu0 + fyu3 + fzu6), (fxu1 + fyu4 + fzu7), (fxu2 + fyu5 + fzu8));
  //mprintf("DEBUG: a1: %g %g %g\n", X_factor, Y_factor, Z_factor);
  double min = (x*x) + (y*y) + (z*z);

  if (minIn > 0.0 && minIn < min) min = minIn;

  //ixyz[0] = 0;
  //ixyz[1] = 0;
  //ixyz[2] = 0;

  // -1 -1 -1
  x = (fxm1u0 + fym1u3 + fzm1u6) - X_factor;
  y = (fxm1u1 + fym1u4 + fzm1u7) - Y_factor;
  z = (fxm1u2 + fym1u5 + fzm1u8) - Z_factor;
  double D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] = -1; ixyz[2] = -1; }
  if (D < min) min = D;
  // -1 -1  0
  x = (fxm1u0 + fym1u3 + fzu6  ) - X_factor;
  y = (fxm1u1 + fym1u4 + fzu7  ) - Y_factor;
  z = (fxm1u2 + fym1u5 + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] = -1; ixyz[2] =  0; }
  if (D < min) min = D;
  // -1 -1 +1
  x = (fxm1u0 + fym1u3 + fzp1u6) - X_factor;
  y = (fxm1u1 + fym1u4 + fzp1u7) - Y_factor;
  z = (fxm1u2 + fym1u5 + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] = -1; ixyz[2] =  1; }
  if (D < min) min = D;
  // -1  0 -1
  x = (fxm1u0 + fyu3   + fzm1u6) - X_factor;
  y = (fxm1u1 + fyu4   + fzm1u7) - Y_factor;
  z = (fxm1u2 + fyu5   + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] =  0; ixyz[2] = -1; }
  if (D < min) min = D;
  // -1  0  0
  x = (fxm1u0 + fyu3   + fzu6  ) - X_factor;
  y = (fxm1u1 + fyu4   + fzu7  ) - Y_factor;
  z = (fxm1u2 + fyu5   + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] =  0; ixyz[2] =  0; }
  if (D < min) min = D;
  // -1  0 +1
  x = (fxm1u0 + fyu3   + fzp1u6) - X_factor;
  y = (fxm1u1 + fyu4   + fzp1u7) - Y_factor;
  z = (fxm1u2 + fyu5   + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] =  0; ixyz[2] =  1; }
  if (D < min) min = D;
  // -1 +1 -1
  x = (fxm1u0 + fyp1u3 + fzm1u6) - X_factor;
  y = (fxm1u1 + fyp1u4 + fzm1u7) - Y_factor;
  z = (fxm1u2 + fyp1u5 + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] =  1; ixyz[2] = -1; }
  if (D < min) min = D;
  // -1 +1  0
  x = (fxm1u0 + fyp1u3 + fzu6  ) - X_factor;
  y = (fxm1u1 + fyp1u4 + fzu7  ) - Y_factor;
  z = (fxm1u2 + fyp1u5 + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] =  1; ixyz[2] =  0; }
  if (D < min) min = D;
  // -1 +1 +1
  x = (fxm1u0 + fyp1u3 + fzp1u6) - X_factor;
  y = (fxm1u1 + fyp1u4 + fzp1u7) - Y_factor;
  z = (fxm1u2 + fyp1u5 + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] = -1; ixyz[1] =  1; ixyz[2] =  1; }
  if (D < min) min = D;

  //  0 -1 -1
  x = (fxu0   + fym1u3 + fzm1u6) - X_factor;
  y = (fxu1   + fym1u4 + fzm1u7) - Y_factor;
  z = (fxu2   + fym1u5 + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] = -1; ixyz[2] = -1; }
  if (D < min) min = D;
  //  0 -1  0
  x = (fxu0   + fym1u3 + fzu6  ) - X_factor;
  y = (fxu1   + fym1u4 + fzu7  ) - Y_factor;
  z = (fxu2   + fym1u5 + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] = -1; ixyz[2] =  0; }
  if (D < min) min = D;
  //  0 -1 +1
  x = (fxu0   + fym1u3 + fzp1u6) - X_factor;
  y = (fxu1   + fym1u4 + fzp1u7) - Y_factor;
  z = (fxu2   + fym1u5 + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] = -1; ixyz[2] =  1; }
  if (D < min) min = D;
  //  0  0 -1
  x = (fxu0   + fyu3   + fzm1u6) - X_factor;
  y = (fxu1   + fyu4   + fzm1u7) - Y_factor;
  z = (fxu2   + fyu5   + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] =  0; ixyz[2] = -1; }
  if (D < min) min = D;
  //  0  0  0
  //  0  0 +1
  x = (fxu0   + fyu3   + fzp1u6) - X_factor;
  y = (fxu1   + fyu4   + fzp1u7) - Y_factor;
  z = (fxu2   + fyu5   + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] =  0; ixyz[2] =  1; }
  if (D < min) min = D;
  //  0 +1 -1
  x = (fxu0   + fyp1u3 + fzm1u6) - X_factor;
  y = (fxu1   + fyp1u4 + fzm1u7) - Y_factor;
  z = (fxu2   + fyp1u5 + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] =  1; ixyz[2] = -1; }
  if (D < min) min = D;
  //  0 +1  0
  x = (fxu0   + fyp1u3 + fzu6  ) - X_factor;
  y = (fxu1   + fyp1u4 + fzu7  ) - Y_factor;
  z = (fxu2   + fyp1u5 + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] =  1; ixyz[2] =  0; }
  if (D < min) min = D;
  //  0 +1 +1
  x = (fxu0   + fyp1u3 + fzp1u6) - X_factor;
  y = (fxu1   + fyp1u4 + fzp1u7) - Y_factor;
  z = (fxu2   + fyp1u5 + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  0; ixyz[1] =  1; ixyz[2] =  1; }
  if (D < min) min = D;

  // +1 -1 -1
  x = (fxp1u0 + fym1u3 + fzm1u6) - X_factor;
  y = (fxp1u1 + fym1u4 + fzm1u7) - Y_factor;
  z = (fxp1u2 + fym1u5 + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] = -1; ixyz[2] = -1; }
  if (D < min) min = D;
  // +1 -1  0
  x = (fxp1u0 + fym1u3 + fzu6  ) - X_factor;
  y = (fxp1u1 + fym1u4 + fzu7  ) - Y_factor;
  z = (fxp1u2 + fym1u5 + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] = -1; ixyz[2] =  0; }
  if (D < min) min = D;
  // +1 -1 +1
  x = (fxp1u0 + fym1u3 + fzp1u6) - X_factor;
  y = (fxp1u1 + fym1u4 + fzp1u7) - Y_factor;
  z = (fxp1u2 + fym1u5 + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] = -1; ixyz[2] =  1; }
  if (D < min) min = D;
  // +1  0 -1
  x = (fxp1u0 + fyu3   + fzm1u6) - X_factor;
  y = (fxp1u1 + fyu4   + fzm1u7) - Y_factor;
  z = (fxp1u2 + fyu5   + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] =  0; ixyz[2] = -1; }
  if (D < min) min = D;
  // +1  0  0
  x = (fxp1u0 + fyu3   + fzu6  ) - X_factor;
  y = (fxp1u1 + fyu4   + fzu7  ) - Y_factor;
  z = (fxp1u2 + fyu5   + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] =  0; ixyz[2] =  0; }
  if (D < min) min = D;
  // +1  0 +1
  x = (fxp1u0 + fyu3   + fzp1u6) - X_factor;
  y = (fxp1u1 + fyu4   + fzp1u7) - Y_factor;
  z = (fxp1u2 + fyu5   + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] =  0; ixyz[2] =  1; }
  if (D < min) min = D;
  // +1 +1 -1
  x = (fxp1u0 + fyp1u3 + fzm1u6) - X_factor;
  y = (fxp1u1 + fyp1u4 + fzm1u7) - Y_factor;
  z = (fxp1u2 + fyp1u5 + fzm1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] =  1; ixyz[2] = -1; }
  if (D < min) min = D;
  // +1 +1  0
  x = (fxp1u0 + fyp1u3 + fzu6  ) - X_factor;
  y = (fxp1u1 + fyp1u4 + fzu7  ) - Y_factor;
  z = (fxp1u2 + fyp1u5 + fzu8  ) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] =  1; ixyz[2] =  0; }
  if (D < min) min = D;
  // +1 +1 +1
  x = (fxp1u0 + fyp1u3 + fzp1u6) - X_factor;
  y = (fxp1u1 + fyp1u4 + fzp1u7) - Y_factor;
  z = (fxp1u2 + fyp1u5 + fzp1u8) - Z_factor;
  D = (x*x) + (y*y) + (z*z);
  //if (D < min) { min = D; ixyz[0] =  1; ixyz[1] =  1; ixyz[2] =  1; }
  if (D < min) min = D;

  //if (closest2 != 0.0 && min < closest2) return (min);
//  this->ClosestImage(a1, a2, ixyz);
//  fprintf(stdout,"DEBUG: Predict  = %2i %2i %2i\n",ixyz[0],ixyz[1],ixyz[2]);

//  ix = ixyz[0];
//  iy = ixyz[1];
//  iz = ixyz[2];

//D = sqrt(min);
//  fprintf(stdout,"DEBUG: MinDist  = %2i %2i %2i = %8.3f\n", ixmin, iymin, izmin, D);
//  printf("---------------------------------------------------------------\n");
  return(min);

}

// -----------------------------------------------------------------------------
//try thread coarsening 
/** Calculate the closest distances of atoms of solvent molecules to a point. */
__global__ void kClosestDistsToPt_NoImage(double* D_, const double* maskCenter,
                                          const double* SolventMols_,
                                          double maxD, int Nmols, int NAtoms,
                                          int active_size)
{
  //__shared__ double dist_array[BLOCKDIM];

  //int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  //int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  int mol = blockIdx.x*BLOCKDIM + threadIdx.x;

  //advantage of register
  double a0 = maskCenter[0];
  double a1 = maskCenter[1];
  double a2 = maskCenter[2];

  if ( mol < Nmols )
  {
    int sIndex =  mol*NAtoms*3;
    double min_val  = maxD;
    for(int offset  = 0 ; offset < NAtoms*3 ; offset+=3 )
    {
      //double x = a0 - SolventMols_[sIndex++];
      //double y = a1 - SolventMols_[sIndex++];
      //double z = a2 - SolventMols_[sIndex++];

      double x = a0 - SolventMols_[sIndex+ offset + 0 ];
      double y = a1 - SolventMols_[sIndex+offset + 1];
      double z = a2 - SolventMols_[sIndex+offset + 2];

      min_val  =  min(min_val, x*x + y*y + z*z);
    }

    D_[mol] = min_val;
  }
}


// -----------------------------------------------------------------------------
/** Calculate closest distances of atoms of solvent molecules to solute atoms.
  */
__global__ void kClosestDistsToAtoms_NoImage(double* D_,
                                             const double* SolventMols_,
                                             const double* Solute_atoms,
                                             double maxD, int Nmols, int NAtoms,
                                             int NSAtoms, int active_size)
{
  __shared__ double dist_array[BLOCKDIM];
  //__shared__ double sAtom_shared[RSIZE];

  int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  //handling the chunks for  solute_atoms
  int chunksize,start,end, NChunks,i,j;

  if(NSAtoms*3 > RSIZE)
  {
    chunksize = (RSIZE/3)*3;
    NChunks = ceil(double(NSAtoms*3)/chunksize);
    start = 0;
    end = chunksize;
  }
  else
  {
    chunksize = NSAtoms*3;
    NChunks = 1;
    start = 0;
    end = NSAtoms*3;
  }

  // if(threadIdx.x == 0 && blockIdx.x == 0 )
  //   printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
  //     chunksize,NChunks,start,end);

  if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
  {
    // if(atom == 0 )
    //   D_[mol] = maxD;
    //__syncthreads(); 
    double min_val  = maxD;
    double dist;
    int sIndex =  mol*NAtoms*3 + atom*3;
    double a0 = SolventMols_[sIndex + 0];
    double a1 = SolventMols_[sIndex + 1];
    double a2 = SolventMols_[sIndex + 2];

    //this is to imporve cache hits! (in the old days this would be thrown in shared mem)
    for(i  = 0 ; i  < NChunks ; i++)
    {
      //copying to shared
      //if (threadIdx.x < (end - start))
      //  sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

      //__syncthreads();

      //TODO - add skew per thread 
      for (j = start ; j < end; j+=3 )
      {
        //int offset = start + (j + threadIdx.x)%(end - start);
        double x = Solute_atoms[j + 0]  - a0;
        double y = Solute_atoms[j + 1]  - a1;
        double z = Solute_atoms[j + 2]  - a2;
        dist =  x*x + y*y + z*z;
        //if (mol ==  11)
        //  printf("min  = %f\n",min_val);
        min_val = min(min_val,dist);


      }

      start = end;
      end = min(end + chunksize, NSAtoms*3);
    }

    dist_array[threadIdx.x] = min_val;
    //if (threadIdx.x == 0)
    //  printf("min_val  = %f\n",min_val);
    //printf(" dist  =  %f\n", Dist);

    __syncthreads();

    //first thread
    //naive approach to a reduction algorithm
    //this works if NAtoms is small other wise you need split
    //and do some of log(n) parallel reduction 
    //min_val  = maxD;
    if( atom ==0 )
    {
      for(i  = 0 ; i < NAtoms ; i++ ){
        //sIndex = mol*NAtoms*3 + i*3;
        //if (dist_array[threadIdx.x + i]  < min_val) 
        //  min_val = dist_array[threadIdx.x + i] ;
        min_val =  min(min_val, dist_array[threadIdx.x + i]);
      }
      D_[mol] = min_val;
    }
  //if(tx == 0 && bx == 0 )
  //  printf("end of kernel");
  }
}

// -----------------------------------------------------------------------------
/** Calculate the closest distances of atoms of solvent molecules to a point.
  * Perform orthorhombic imaging.
  */
__global__ void kClosestDistsToPt_Ortho(double *D_, const double* maskCenter,
                                        const double* SolventMols_, double maxD,
                                        const double *box, int Nmols,
                                        int NAtoms, int active_size)
{
  //__shared__ double dist_array[BLOCKDIM];

  //int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  //int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  int mol = blockIdx.x*BLOCKDIM + threadIdx.x;

  //advantage of register
  double a0 = maskCenter[0];
  double a1 = maskCenter[1];
  double a2 = maskCenter[2];

  if ( mol < Nmols )
  {
    int sIndex =  mol*NAtoms*3;
    double min_val  = maxD;
    double dist;
    for(int offset  = 0 ; offset < NAtoms ; offset++ )
    {
      double x = a0 - SolventMols_[sIndex++];
      double y = a1 - SolventMols_[sIndex++];
      double z = a2 - SolventMols_[sIndex++];

      // Get rid of sign info
      if (x<0) x=-x;
      if (y<0) y=-y;
      if (z<0) z=-z;
      // Get rid of multiples of box lengths 
      //TODO  WIERD that should be a way to simplify it
      while (x > box[0]) x = x - box[0];
      while (y > box[1]) y = y - box[1];
      while (z > box[2]) z = z - box[2];
        // Find shortest distance in periodic reference
      double D = box[0] - x;
      if (D < x) x = D;
      D = box[1] - y;
      if (D < y) y = D;  
      D = box[2] - z;
      if (D < z) z = D;

      //Dist = x*x + y*y + z*z;
      dist = x*x + y*y + z*z;
      if (box[0]==0.0 || box[1]==0.0 || box[2]==0.0)
        dist= -1.0;

      min_val  =  min(min_val, dist);
    }

    D_[mol] = min_val;
  }
}

//------------------------------------------------------------------------------
/** Calculate closest distances of atoms of solvent molecules to solute atoms.
  * Perform orthorhombic imaging.
  */
__global__ void kClosestDistsToAtoms_Ortho(double* D_, const double* SolventMols_,
                                           const double* Solute_atoms, double maxD,
                                           const double* box, int Nmols, int NAtoms,
                                           int NSAtoms, int active_size)
{
  __shared__ double dist_array[BLOCKDIM];
  //__shared__ double sAtom_shared[RSIZE];

  int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  //handling the chunks for  solute_atoms
  int chunksize,start,end, NChunks,i,j;

  if(NSAtoms*3 > RSIZE)
  {
    chunksize = (RSIZE/3)*3;
    NChunks = ceil(double(NSAtoms*3)/chunksize);
    start = 0;
    end = chunksize;
  }
  else
  {
    chunksize = NSAtoms*3;
    NChunks = 1;
    start = 0;
    end = NSAtoms*3;
  }

  // if(threadIdx.x == 0 && blockIdx.x == 0 )
  //   printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
  //     chunksize,NChunks,start,end);

  if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
  {
    // if(atom == 0 )
    //   D_[mol] = maxD;
    //__syncthreads(); 
    double min_val  = maxD;
    double dist;
    int sIndex =  mol*NAtoms*3 + atom*3;
    double a0 = SolventMols_[sIndex + 0];
    double a1 = SolventMols_[sIndex + 1];
    double a2 = SolventMols_[sIndex + 2];

    for(i  = 0 ; i  < NChunks ; i++)
    {
      //copying to shared
      //if (threadIdx.x < (end - start))
      //  sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

      //__syncthreads();

      //TODO - add skew per thread 
      for (j = start ; j < end; j+=3 )
      {
        //int offset = start + (j + threadIdx.x)%(end - start);
        double x = Solute_atoms[j + 0]  - a0;
        double y = Solute_atoms[j + 1]  - a1;
        double z = Solute_atoms[j + 2]  - a2;

        // Get rid of sign info
        if (x<0) x=-x;
        if (y<0) y=-y;
        if (z<0) z=-z;
        // Get rid of multiples of box lengths 
        //TODO  WIERD that should be a way to simplify it
        while (x > box[0]) x = x - box[0];
        while (y > box[1]) y = y - box[1];
        while (z > box[2]) z = z - box[2];

        //below is actually slower! 
        //x = x - box[0]*((int)x/box[0]);
        //y = y - box[0]*((int)y/box[1]);
        //z = z - box[0]*((int)z/box[2]);
        // Find shortest distance in periodic reference
        double D = box[0] - x;
        if (D < x) x = D;
        D = box[1] - y;
        if (D < y) y = D;  
        D = box[2] - z;
        if (D < z) z = D;

        //Dist = x*x + y*y + z*z;
        dist =  x*x + y*y + z*z;
        if (box[0]==0.0 || box[1]==0.0 || box[2]==0.0)
          dist = -1.0;

        //if (mol ==  11)
        //  printf("min  = %f\n",min_val);
        min_val = min(min_val,dist);
      }

      start = end;
      end = min(end + chunksize, NSAtoms*3);
    }

    dist_array[threadIdx.x] = min_val;
    //if (threadIdx.x == 0)
    //  printf("min_val  = %f\n",min_val);
    //printf(" dist  =  %f\n", Dist);

    __syncthreads();

    //first thread
    //naive approach to a reduction algorithm
    //this works if NAtoms is small other wise you need split
    //and do some of log(n) parallel reduction 
    //min_val  = maxD;
    if( atom ==0 )
    {
      for(i  = 0 ; i < NAtoms ; i++ ){
        //sIndex = mol*NAtoms*3 + i*3;
        //if (dist_array[threadIdx.x + i]  < min_val) 
        //  min_val = dist_array[threadIdx.x + i] ;
        min_val =  min(min_val, dist_array[threadIdx.x + i]);
      }
      D_[mol] = min_val;
    }

  //if(tx == 0 && bx == 0 )
  //  printf("end of kernel");
  }
}

// -----------------------------------------------------------------------------
/** Calculate the closest distances of atoms of solvent molecules to a point.
  * Perform non-orthorhombic imaging.
  */
__global__ void kClosestDistsToPt_Nonortho(double* D_, const double* maskCenter,
                                           const double* SolventMols_,
                                           double maxD, const double* ucell,
                                           const double *recip, int Nmols,
                                           int NAtoms, int active_size)
{
  //__shared__ double dist_array[BLOCKDIM];

  //int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  //int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  int mol = blockIdx.x*BLOCKDIM + threadIdx.x;

  //advantage of register
  double a0 = recip[0]*maskCenter[0] + recip[1]*maskCenter[1] + recip[2]*maskCenter[2];
  double a1 = recip[3]*maskCenter[0] + recip[4]*maskCenter[1] + recip[5]*maskCenter[2];
  double a2 = recip[6]*maskCenter[0] + recip[7]*maskCenter[1] + recip[8]*maskCenter[2];

  if ( mol < Nmols )
  {
    int sIndex =  mol*NAtoms*3;
    double min_val  = maxD;
    for(int offset  = 0 ; offset < NAtoms*3 ; offset+=3 )
    {
      double x =  recip[0]*SolventMols_[sIndex + offset + 0] + recip[1]*SolventMols_[sIndex + offset + 1] + recip[2]*SolventMols_[sIndex + offset + 2];
      double y =  recip[3]*SolventMols_[sIndex + offset + 0] + recip[4]*SolventMols_[sIndex + offset + 1] + recip[5]*SolventMols_[sIndex + offset + 2];
      double z =  recip[6]*SolventMols_[sIndex + offset + 0] + recip[7]*SolventMols_[sIndex + offset + 1] + recip[8]*SolventMols_[sIndex + offset + 2];
      double dist  = NonOrtho_dist(x,y,z,a0,a1,a2,ucell);
      // if (mol ==  0)
      //   printf("dist  = %f\n",dist);

      min_val  =  min(min_val, dist);
    }

    D_[mol] = min_val;
  }
}

// -----------------------------------------------------------------------------
/** Calculate closest distances of atoms of solvent molecules to solute atoms.
  * Perform non-orthorhombic imaging.
  */
__global__ void kClosestDistsToAtoms_Nonortho(double*D_,
                                              const double* SolventMols_,
                                              const double* Solute_atoms,
                                              double maxD, const double *ucell,
                                              const double* recip, int Nmols,
                                              int NAtoms, int NSAtoms,
                                              int active_size)
{
  __shared__ double dist_array[BLOCKDIM];
  //__shared__ double sAtom_shared[RSIZE];

  int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  //handling the chunks for  solute_atoms
  int chunksize,start,end, NChunks,i,j;

  if(NSAtoms*3 > RSIZE)
  {
    chunksize = (RSIZE/3)*3;
    NChunks = ceil(double(NSAtoms*3)/chunksize);
    start = 0;
    end = chunksize;
  }
  else
  {
    chunksize = NSAtoms*3;
    NChunks = 1;
    start = 0;
    end = NSAtoms*3;
  }

  // if(threadIdx.x == 0 && blockIdx.x == 0 )
  //   printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
  //     chunksize,NChunks,start,end);

  if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
  {
    // if(atom == 0 )
    //   D_[mol] = maxD;
    //__syncthreads(); 
    double min_val  = maxD;
    double dist;
    int sIndex =  mol*NAtoms*3 + atom*3;

    double a0 = recip[0]*SolventMols_[sIndex + 0] + recip[1]*SolventMols_[sIndex + 1] + recip[2]*SolventMols_[sIndex + 2];
    double a1 = recip[3]*SolventMols_[sIndex + 0] + recip[4]*SolventMols_[sIndex + 1] + recip[5]*SolventMols_[sIndex + 2];
    double a2 = recip[6]*SolventMols_[sIndex + 0] + recip[7]*SolventMols_[sIndex + 1] + recip[8]*SolventMols_[sIndex + 2];

    for(i  = 0 ; i  < NChunks ; i++)
    {
      //copying to shared
      //if (threadIdx.x < (end - start))
      //  sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

      //__syncthreads();

      //TODO - add skew per thread 
      for (j = start ; j < end; j+=3 )
      {
        //int offset = start + (j + threadIdx.x)%(end - start);

        double x = recip[0]*Solute_atoms[j + 0]  + recip[1]*Solute_atoms[j + 1]  + recip[2]*Solute_atoms[j + 2] ;
        double y = recip[3]*Solute_atoms[j + 0]  + recip[4]*Solute_atoms[j + 1]  + recip[5]*Solute_atoms[j + 2] ;
        double z = recip[6]*Solute_atoms[j + 0]  + recip[7]*Solute_atoms[j + 1]  + recip[8]*Solute_atoms[j + 2] ;

        dist =  NonOrtho_dist(x,y,z,a0,a1,a2,ucell);
        //if (mol ==  11)
        //  printf("min  = %f\n",min_val);
        min_val = min(min_val,dist);
      }

      start = end;
      end = min(end + chunksize, NSAtoms*3);
    }

    dist_array[threadIdx.x] = min_val;
    //if (threadIdx.x == 0)
    //  printf("min_val  = %f\n",min_val);
    //printf(" dist  =  %f\n", Dist);

    __syncthreads();

    //first thread
    //naive approach to a reduction algorithm
    //this works if NAtoms is small other wise you need split
    //and do some of log(n) parallel reduction 
    //min_val  = maxD;
    if( atom ==0 )
    {
      for(i  = 0 ; i < NAtoms ; i++ ){
        //sIndex = mol*NAtoms*3 + i*3;
        //if (dist_array[threadIdx.x + i]  < min_val) 
        //  min_val = dist_array[threadIdx.x + i] ;
        min_val =  min(min_val, dist_array[threadIdx.x + i]);
      }
      D_[mol] = min_val;
    }

  //if(tx == 0 && bx == 0 )
  //  printf("end of kernel");
  }
}


