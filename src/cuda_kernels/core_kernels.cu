#include "hip/hip_runtime.h"
#include "core_kernels.cuh"
#include "NonOrtho_dist2.cuh"
#include "ortho_dist2.cuh"
//#include <cstdio> // DEBUG
#define BLOCKDIM 512
#define RSIZE 512

// -----------------------------------------------------------------------------
//try thread coarsening 
/** Calculate the closest distances of atoms of solvent molecules to a point. */
__global__ void kClosestDistsToPt_NoImage(double* D_, const double* maskCenter,
                                          const double* SolventMols_,
                                          double maxD, int Nmols, int NAtoms,
                                          int active_size)
{
  //__shared__ double dist_array[BLOCKDIM];

  //int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  //int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  int mol = blockIdx.x*BLOCKDIM + threadIdx.x;

  //advantage of register
  double a0 = maskCenter[0];
  double a1 = maskCenter[1];
  double a2 = maskCenter[2];

  if ( mol < Nmols )
  {
    int sIndex =  mol*NAtoms*3;
    double min_val  = maxD;
    for(int offset  = 0 ; offset < NAtoms*3 ; offset+=3 )
    {
      //double x = a0 - SolventMols_[sIndex++];
      //double y = a1 - SolventMols_[sIndex++];
      //double z = a2 - SolventMols_[sIndex++];

      double x = a0 - SolventMols_[sIndex+ offset + 0 ];
      double y = a1 - SolventMols_[sIndex+offset + 1];
      double z = a2 - SolventMols_[sIndex+offset + 2];

      min_val  =  min(min_val, x*x + y*y + z*z);
    }

    D_[mol] = min_val;
  }
}

// -----------------------------------------------------------------------------
/** Calculate closest distances of atoms of solvent molecules to solute atoms.
  */
__global__ void kClosestDistsToAtoms_NoImage(double* D_,
                                             const double* SolventMols_,
                                             const double* Solute_atoms,
                                             double maxD, int Nmols, int NAtoms,
                                             int NSAtoms, int active_size)
{
  __shared__ double dist_array[BLOCKDIM];
  //__shared__ double sAtom_shared[RSIZE];

  int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  //handling the chunks for  solute_atoms
  int chunksize,start,end, NChunks,i,j;

  if(NSAtoms*3 > RSIZE)
  {
    chunksize = (RSIZE/3)*3;
    NChunks = ceil(double(NSAtoms*3)/chunksize);
    start = 0;
    end = chunksize;
  }
  else
  {
    chunksize = NSAtoms*3;
    NChunks = 1;
    start = 0;
    end = NSAtoms*3;
  }

  // if(threadIdx.x == 0 && blockIdx.x == 0 )
  //   printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
  //     chunksize,NChunks,start,end);

  if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
  {
    // if(atom == 0 )
    //   D_[mol] = maxD;
    //__syncthreads(); 
    double min_val  = maxD;
    double dist;
    int sIndex =  mol*NAtoms*3 + atom*3;
    double a0 = SolventMols_[sIndex + 0];
    double a1 = SolventMols_[sIndex + 1];
    double a2 = SolventMols_[sIndex + 2];

    //this is to imporve cache hits! (in the old days this would be thrown in shared mem)
    for(i  = 0 ; i  < NChunks ; i++)
    {
      //copying to shared
      //if (threadIdx.x < (end - start))
      //  sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

      //__syncthreads();

      //TODO - add skew per thread 
      for (j = start ; j < end; j+=3 )
      {
        //int offset = start + (j + threadIdx.x)%(end - start);
        double x = Solute_atoms[j + 0]  - a0;
        double y = Solute_atoms[j + 1]  - a1;
        double z = Solute_atoms[j + 2]  - a2;
        dist =  x*x + y*y + z*z;
        //if (mol ==  11)
        //  printf("min  = %f\n",min_val);
        min_val = min(min_val,dist);


      }

      start = end;
      end = min(end + chunksize, NSAtoms*3);
    }

    dist_array[threadIdx.x] = min_val;
    //if (threadIdx.x == 0)
    //  printf("min_val  = %f\n",min_val);
    //printf(" dist  =  %f\n", Dist);

    __syncthreads();

    //first thread
    //naive approach to a reduction algorithm
    //this works if NAtoms is small other wise you need split
    //and do some of log(n) parallel reduction 
    //min_val  = maxD;
    if( atom ==0 )
    {
      for(i  = 0 ; i < NAtoms ; i++ ){
        //sIndex = mol*NAtoms*3 + i*3;
        //if (dist_array[threadIdx.x + i]  < min_val) 
        //  min_val = dist_array[threadIdx.x + i] ;
        min_val =  min(min_val, dist_array[threadIdx.x + i]);
      }
      D_[mol] = min_val;
    }
  //if(tx == 0 && bx == 0 )
  //  printf("end of kernel");
  }
}

// -----------------------------------------------------------------------------
/** Calculate the closest distances of atoms of solvent molecules to a point.
  * Perform orthorhombic imaging.
  */
__global__ void kClosestDistsToPt_Ortho(double *D_, const double* maskCenter,
                                        const double* SolventMols_, double maxD,
                                        const double *box, int Nmols,
                                        int NAtoms, int active_size)
{
  //__shared__ double dist_array[BLOCKDIM];

  //int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  //int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  int mol = blockIdx.x*BLOCKDIM + threadIdx.x;

  //advantage of register
  double a0 = maskCenter[0];
  double a1 = maskCenter[1];
  double a2 = maskCenter[2];

  if ( mol < Nmols )
  {
    int sIndex =  mol*NAtoms*3;
    double min_val  = maxD;
    double dist;
    for(int offset  = 0 ; offset < NAtoms ; offset++ )
    {
      dist = ortho_dist2<double>( a0, a1, a2,
                                  SolventMols_[sIndex], SolventMols_[sIndex+1], SolventMols_[sIndex+2],
                                  box );
      sIndex += 3;

      if (box[0]==0.0 || box[1]==0.0 || box[2]==0.0)
        dist= -1.0;

      min_val  =  min(min_val, dist);
    }

    D_[mol] = min_val;
  }
}

//------------------------------------------------------------------------------
/** Calculate closest distances of atoms of solvent molecules to solute atoms.
  * Perform orthorhombic imaging.
  */
__global__ void kClosestDistsToAtoms_Ortho(double* D_, const double* SolventMols_,
                                           const double* Solute_atoms, double maxD,
                                           const double* box, int Nmols, int NAtoms,
                                           int NSAtoms, int active_size)
{
  __shared__ double dist_array[BLOCKDIM];
  //__shared__ double sAtom_shared[RSIZE];

  int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  //handling the chunks for  solute_atoms
  int chunksize,start,end, NChunks,i,j;

  if(NSAtoms*3 > RSIZE)
  {
    chunksize = (RSIZE/3)*3;
    NChunks = ceil(double(NSAtoms*3)/chunksize);
    start = 0;
    end = chunksize;
  }
  else
  {
    chunksize = NSAtoms*3;
    NChunks = 1;
    start = 0;
    end = NSAtoms*3;
  }

  // if(threadIdx.x == 0 && blockIdx.x == 0 )
  //   printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
  //     chunksize,NChunks,start,end);

  if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
  {
    // if(atom == 0 )
    //   D_[mol] = maxD;
    //__syncthreads(); 
    double min_val  = maxD;
    double dist;
    int sIndex =  mol*NAtoms*3 + atom*3;
    double a0 = SolventMols_[sIndex + 0];
    double a1 = SolventMols_[sIndex + 1];
    double a2 = SolventMols_[sIndex + 2];

    for(i  = 0 ; i  < NChunks ; i++)
    {
      //copying to shared
      //if (threadIdx.x < (end - start))
      //  sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

      //__syncthreads();

      //TODO - add skew per thread 
      for (j = start ; j < end; j+=3 )
      {
        //int offset = start + (j + threadIdx.x)%(end - start);
        dist = ortho_dist2<double>( Solute_atoms[j], Solute_atoms[j+1], Solute_atoms[j+2],
                                    a0, a1, a2,
                                    box );

        if (box[0]==0.0 || box[1]==0.0 || box[2]==0.0)
          dist = -1.0;

        //if (mol ==  11)
        //  printf("min  = %f\n",min_val);
        min_val = min(min_val,dist);
      }

      start = end;
      end = min(end + chunksize, NSAtoms*3);
    }

    dist_array[threadIdx.x] = min_val;
    //if (threadIdx.x == 0)
    //  printf("min_val  = %f\n",min_val);
    //printf(" dist  =  %f\n", Dist);

    __syncthreads();

    //first thread
    //naive approach to a reduction algorithm
    //this works if NAtoms is small other wise you need split
    //and do some of log(n) parallel reduction 
    //min_val  = maxD;
    if( atom ==0 )
    {
      for(i  = 0 ; i < NAtoms ; i++ ){
        //sIndex = mol*NAtoms*3 + i*3;
        //if (dist_array[threadIdx.x + i]  < min_val) 
        //  min_val = dist_array[threadIdx.x + i] ;
        min_val =  min(min_val, dist_array[threadIdx.x + i]);
      }
      D_[mol] = min_val;
    }

  //if(tx == 0 && bx == 0 )
  //  printf("end of kernel");
  }
}

// -----------------------------------------------------------------------------
/** Calculate the closest distances of atoms of solvent molecules to a point.
  * Perform non-orthorhombic imaging.
  */
__global__ void kClosestDistsToPt_Nonortho(double* D_, const double* maskCenter,
                                           const double* SolventMols_,
                                           double maxD, const double* ucell,
                                           const double *recip, int Nmols,
                                           int NAtoms, int active_size)
{
  //__shared__ double dist_array[BLOCKDIM];

  //int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  //int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  int mol = blockIdx.x*BLOCKDIM + threadIdx.x;

  //advantage of register
  double a0 = recip[0]*maskCenter[0] + recip[1]*maskCenter[1] + recip[2]*maskCenter[2];
  double a1 = recip[3]*maskCenter[0] + recip[4]*maskCenter[1] + recip[5]*maskCenter[2];
  double a2 = recip[6]*maskCenter[0] + recip[7]*maskCenter[1] + recip[8]*maskCenter[2];

  if ( mol < Nmols )
  {
    int sIndex =  mol*NAtoms*3;
    double min_val  = maxD;
    for(int offset  = 0 ; offset < NAtoms*3 ; offset+=3 )
    {
      double x =  recip[0]*SolventMols_[sIndex + offset + 0] + recip[1]*SolventMols_[sIndex + offset + 1] + recip[2]*SolventMols_[sIndex + offset + 2];
      double y =  recip[3]*SolventMols_[sIndex + offset + 0] + recip[4]*SolventMols_[sIndex + offset + 1] + recip[5]*SolventMols_[sIndex + offset + 2];
      double z =  recip[6]*SolventMols_[sIndex + offset + 0] + recip[7]*SolventMols_[sIndex + offset + 1] + recip[8]*SolventMols_[sIndex + offset + 2];
      double dist  = NonOrtho_dist2<double>(x,y,z,a0,a1,a2,ucell);
      // if (mol ==  0)
      //   printf("dist  = %f\n",dist);

      min_val  =  min(min_val, dist);
    }

    D_[mol] = min_val;
  }
}

// -----------------------------------------------------------------------------
/** Calculate closest distances of atoms of solvent molecules to solute atoms.
  * Perform non-orthorhombic imaging.
  */
__global__ void kClosestDistsToAtoms_Nonortho(double*D_,
                                              const double* SolventMols_,
                                              const double* Solute_atoms,
                                              double maxD, const double *ucell,
                                              const double* recip, int Nmols,
                                              int NAtoms, int NSAtoms,
                                              int active_size)
{
  __shared__ double dist_array[BLOCKDIM];
  //__shared__ double sAtom_shared[RSIZE];

  int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
  int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
  //int mol_in_block = threadIdx.x/NAtoms;

  //handling the chunks for  solute_atoms
  int chunksize,start,end, NChunks,i,j;

  if(NSAtoms*3 > RSIZE)
  {
    chunksize = (RSIZE/3)*3;
    NChunks = ceil(double(NSAtoms*3)/chunksize);
    start = 0;
    end = chunksize;
  }
  else
  {
    chunksize = NSAtoms*3;
    NChunks = 1;
    start = 0;
    end = NSAtoms*3;
  }

  // if(threadIdx.x == 0 && blockIdx.x == 0 )
  //   printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
  //     chunksize,NChunks,start,end);

  if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
  {
    // if(atom == 0 )
    //   D_[mol] = maxD;
    //__syncthreads(); 
    double min_val  = maxD;
    double dist;
    int sIndex =  mol*NAtoms*3 + atom*3;

    double a0 = recip[0]*SolventMols_[sIndex + 0] + recip[1]*SolventMols_[sIndex + 1] + recip[2]*SolventMols_[sIndex + 2];
    double a1 = recip[3]*SolventMols_[sIndex + 0] + recip[4]*SolventMols_[sIndex + 1] + recip[5]*SolventMols_[sIndex + 2];
    double a2 = recip[6]*SolventMols_[sIndex + 0] + recip[7]*SolventMols_[sIndex + 1] + recip[8]*SolventMols_[sIndex + 2];

    for(i  = 0 ; i  < NChunks ; i++)
    {
      //copying to shared
      //if (threadIdx.x < (end - start))
      //  sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

      //__syncthreads();

      //TODO - add skew per thread 
      for (j = start ; j < end; j+=3 )
      {
        //int offset = start + (j + threadIdx.x)%(end - start);

        double x = recip[0]*Solute_atoms[j + 0]  + recip[1]*Solute_atoms[j + 1]  + recip[2]*Solute_atoms[j + 2] ;
        double y = recip[3]*Solute_atoms[j + 0]  + recip[4]*Solute_atoms[j + 1]  + recip[5]*Solute_atoms[j + 2] ;
        double z = recip[6]*Solute_atoms[j + 0]  + recip[7]*Solute_atoms[j + 1]  + recip[8]*Solute_atoms[j + 2] ;

        dist =  NonOrtho_dist2<double>(x,y,z,a0,a1,a2,ucell);
        //if (mol ==  11)
        //  printf("min  = %f\n",min_val);
        min_val = min(min_val,dist);
      }

      start = end;
      end = min(end + chunksize, NSAtoms*3);
    }

    dist_array[threadIdx.x] = min_val;
    //if (threadIdx.x == 0)
    //  printf("min_val  = %f\n",min_val);
    //printf(" dist  =  %f\n", Dist);

    __syncthreads();

    //first thread
    //naive approach to a reduction algorithm
    //this works if NAtoms is small other wise you need split
    //and do some of log(n) parallel reduction 
    //min_val  = maxD;
    if( atom ==0 )
    {
      for(i  = 0 ; i < NAtoms ; i++ ){
        //sIndex = mol*NAtoms*3 + i*3;
        //if (dist_array[threadIdx.x + i]  < min_val) 
        //  min_val = dist_array[threadIdx.x + i] ;
        min_val =  min(min_val, dist_array[threadIdx.x + i]);
      }
      D_[mol] = min_val;
    }

  //if(tx == 0 && bx == 0 )
  //  printf("end of kernel");
  }
}

// -----------------------------------------------------------------------------
/** Bin distances from two non-overlapping sets of coords. */
__global__ void kBinDistances_nonOverlap_NoImage(int* RDF,
                                               const double* xyz1, int N1, const double* xyz2, int N2,
                                               double maximum2, double one_over_spacing)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    double a1x = xyz1[idx1  ];
    double a1y = xyz1[idx1+1];
    double a1z = xyz1[idx1+2];

    int idx2 = a2 * 3;
    double x = a1x - xyz2[idx2  ];
    double y = a1y - xyz2[idx2+1];
    double z = a1z - xyz2[idx2+2];

    double dist2 = (x*x) + (y*y) + (z*z); 
    if (dist2 > 0 && dist2 <= maximum2) {
      double dist = sqrt(dist2);
      int histIdx = (int) (dist * one_over_spacing);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i\n", a1+1, a2+1, dist, histIdx);
      //printf("DEBUG: xyz1= %f %f %f\n", a1x, a1y, a1z);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i  xyz1=%f %f %f  xyz2=%f %f %f\n", a1+1, a2+1, dist, histIdx,
      //       a1x, a1y, a1z, a2x, a2y, a2z);
      atomicAdd( RDF + histIdx, 1 );
    }
  }
}

/** Bin distances from two non-overlapping sets of coords. */
__global__ void kBinDistances_nonOverlap_Ortho(int* RDF,
                                               const double* xyz1, int N1, const double* xyz2, int N2,
                                               const double* box,
                                               double maximum2, double one_over_spacing)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    double a1x = xyz1[idx1  ];
    double a1y = xyz1[idx1+1];
    double a1z = xyz1[idx1+2];

    int idx2 = a2 * 3;
    double a2x = xyz2[idx2  ];
    double a2y = xyz2[idx2+1];
    double a2z = xyz2[idx2+2];

    double dist2 = ortho_dist2<double>(a1x, a1y, a1z, a2x, a2y, a2z, box);
    if (dist2 > 0 && dist2 <= maximum2) {
      double dist = sqrt(dist2);
      int histIdx = (int) (dist * one_over_spacing);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i\n", a1+1, a2+1, dist, histIdx);
      //printf("DEBUG: xyz1= %f %f %f\n", a1x, a1y, a1z);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i  xyz1=%f %f %f  xyz2=%f %f %f\n", a1+1, a2+1, dist, histIdx,
      //       a1x, a1y, a1z, a2x, a2y, a2z);
      atomicAdd( RDF + histIdx, 1 );
    }
  }
}

/** Bin distances from two non-overlapping sets of coords. */
__global__ void kBinDistances_nonOverlap_nonOrtho(int* RDF,
                                                  const double* xyz1, int N1, const double* xyz2, int N2,
                                                  const double* frac, const double* ucell,
                                                  double maximum2, double one_over_spacing)
{
  int a1 = blockIdx.x * blockDim.x + threadIdx.x;
  int a2 = blockIdx.y * blockDim.y + threadIdx.y;

  if (a1 < N1 && a2 < N2) {
    int idx1 = a1 * 3;
    double a1x = xyz1[idx1  ];
    double a1y = xyz1[idx1+1];
    double a1z = xyz1[idx1+2];
    double f1x = frac[0]*a1x + frac[1]*a1y + frac[2]*a1z;
    double f1y = frac[3]*a1x + frac[4]*a1y + frac[5]*a1z;
    double f1z = frac[6]*a1x + frac[7]*a1y + frac[8]*a1z;

    int idx2 = a2 * 3;
    double a2x = xyz2[idx2  ];
    double a2y = xyz2[idx2+1];
    double a2z = xyz2[idx2+2];
    double f2x = frac[0]*a2x + frac[1]*a2y + frac[2]*a2z;
    double f2y = frac[3]*a2x + frac[4]*a2y + frac[5]*a2z;
    double f2z = frac[6]*a2x + frac[7]*a2y + frac[8]*a2z;

    double dist2 =  NonOrtho_dist2<double>(f2x, f2y, f2z, f1x ,f1y, f1z, ucell);
    if (dist2 > 0 && dist2 <= maximum2) {
      double dist = sqrt(dist2);
      int histIdx = (int) (dist * one_over_spacing);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i\n", a1+1, a2+1, dist, histIdx);
      //printf("DEBUG: xyz1= %f %f %f\n", a1x, a1y, a1z);
      //printf("DEBUG: a1= %i  a2= %i  dist= %f  bin=%i  xyz1=%f %f %f  xyz2=%f %f %f\n", a1+1, a2+1, dist, histIdx,
      //       a1x, a1y, a1z, a2x, a2y, a2z);
      atomicAdd( RDF + histIdx, 1 );
    }
  }
}
